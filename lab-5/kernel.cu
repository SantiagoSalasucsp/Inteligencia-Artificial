#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

#define NUM_NEURONAS 10 
#define NUM_ENTRADAS 25      
#define NUM_PATRONES 10 //10 numeros a analizar (0-9)     
#define UMBRAL     0.5f      
#define TASA_APRENDIZAJE 1.0f
#define BIAS_ENTRADA 1.0f

// Para obtener por ejemplo el numero 0, con los 10 numeros(0-9) se debe analizar los pesos(NUM_ENTRADAS) de la primera neurona para su vector sea (1,0,0,0,0,0,0,0,0,0)  
// Para obtener por ejemplo el numero 1, con los 10 numeros(0-9) se debe analizar los pesos(NUM_ENTRADAS) de la segunda neurona para su vector sea (0,1,0,0,0,0,0,0,0,0)  
// Teniendo la matriz de pesos (NUM_NEURONAS*NUM_ENTRADAS)
// En el GPU se debe analizar por threads los vectores de las neuronas. Ya que cada neurona posee una matriz unica de pesos independiente a las demas neuronas. Pudiendo manejarse por thread la actualizacion del vector de pesos de una neurona
//el bias tambien tiene su vector de pesos y deben aplicarse a la formula con su actualizacion de pesos correspondiente.

float entradasCPU[NUM_PATRONES][NUM_ENTRADAS] = {
    // 0
    {1,1,1,1,1,
    1,0,0,0,1,
    1,0,0,0,1,
    1,0,0,0,1,
    1,1,1,1,1},
     // 1
    {0,0,1,0,0,
    0,0,1,0,0,
    0,0,1,0,0,
    0,0,1,0,0,
    0,0,1,0,0},
    // 2
    {1,1,1,1,1,
    0,0,0,0,1,
    1,1,1,1,1,
    1,0,0,0,0,
    1,1,1,1,1},
    // 3
    {1,1,1,1,1,
    0,0,0,0,1,
    0,1,1,1,1,
    0,0,0,0,1,
    1,1,1,1,1},
    // 4
    {1,0,0,0,1,
    1,0,0,0,1,
    1,1,1,1,1,
    0,0,0,0,1,
    0,0,0,0,1},
    // 5
    {1,1,1,1,1,
    1,0,0,0,0,
    1,1,1,1,1,
    0,0,0,0,1,
    1,1,1,1,1},
    // 6
    {1,1,1,1,1,
    1,0,0,0,0,
    1,1,1,1,1,
    1,0,0,0,1,
    1,1,1,1,1},
    // 7
    {1,1,1,1,1,
    0,0,0,0,1,
    0,0,0,0,1,
    0,0,0,0,1,
    0,0,0,0,1},
    // 8
    {1,1,1,1,1,
    1,0,0,0,1,
    1,1,1,1,1,
    1,0,0,0,1,
    1,1,1,1,1},
    // 9
    {1,1,1,1,1,
    1,0,0,0,1,
    1,1,1,1,1,
    0,0,0,0,1,
    1,1,1,1,1}
};

float deseadoCPU[NUM_PATRONES][NUM_NEURONAS] = {
    {1,0,0,0,0,0,0,0,0,0}, // 0
    {0,1,0,0,0,0,0,0,0,0}, // 1
    {0,0,1,0,0,0,0,0,0,0}, // 2
    {0,0,0,1,0,0,0,0,0,0}, // 3
    {0,0,0,0,1,0,0,0,0,0}, // 4
    {0,0,0,0,0,1,0,0,0,0}, // 5
    {0,0,0,0,0,0,1,0,0,0}, // 6
    {0,0,0,0,0,0,0,1,0,0}, // 7
    {0,0,0,0,0,0,0,0,1,0}, // 8
    {0,0,0,0,0,0,0,0,0,1}  // 9
};

// Variables GPU
float* entradasGPU = nullptr; 
float* deseadoGPU = nullptr; 
float* pesosGPU = nullptr; 
float* pesosBiasGPU = nullptr; 
int* errorGPU = nullptr; // un indicador 0 = sin error, 1 = actualizar pesos

// Calcula la salida de las 10 neuronas para un patrón dado y actualiza los pesos
__global__ void entrenamientoGPU(int patronIdx, float* pesosGPU, float* pesosBiasGPU,
    float* entradasGPU, float* deseadoGPU, int* errorGPU) {
    int i = threadIdx.x;  

    // suma de pesos*entredas
    float suma = pesosBiasGPU[i] * BIAS_ENTRADA;
    int offset = patronIdx * NUM_ENTRADAS;  // indice del inicio del patron en el array de entradas
    for (int j = 0; j < NUM_ENTRADAS; ++j) {
        suma += pesosGPU[i * NUM_ENTRADAS + j] * entradasGPU[offset + j];
    }
    float salida = (suma >= UMBRAL) ? 1.0f : 0.0f; //redondeo de la salida

    // Ver si hay error con la igualdad en el deseado y la salida obtenida
    float error = deseadoGPU[patronIdx * NUM_NEURONAS + i] - salida;
    if (error != 0.0f) {
        *errorGPU = 1;
        // actulizar los pesos de la neurona i
        for (int j = 0; j < NUM_ENTRADAS; ++j) {
            // peso_nuevo = peso_viejo + tasa * error * entrada
            pesosGPU[i * NUM_ENTRADAS + j] += TASA_APRENDIZAJE * error * entradasGPU[offset + j];
        }
        // Actualizar el peso del bias de la neurona i
        pesosBiasGPU[i] += TASA_APRENDIZAJE * error * BIAS_ENTRADA;
    }
}

// calcula la salida de las 10 neuronas con los pesos actualizados
__global__ void calcularSumaGPU(int patronIdx, float* pesosGPU, float* pesosBiasGPU,
    float* entradasGPU, float* salidasGPU) {
    int i = threadIdx.x;  
    float suma = pesosBiasGPU[i] * BIAS_ENTRADA;
    int offset = patronIdx * NUM_ENTRADAS;
    for (int j = 0; j < NUM_ENTRADAS; ++j) {
        suma += pesosGPU[i * NUM_ENTRADAS + j] * entradasGPU[offset + j];
    }
    salidasGPU[patronIdx * NUM_NEURONAS + i] = (suma > UMBRAL) ? 1.0f : 0.0f;
}

int main() {
    // ======================================================================================================
    // reservar memoria en GPU 
    hipMalloc((void**)&entradasGPU, NUM_PATRONES * NUM_ENTRADAS * sizeof(float));
    hipMalloc((void**)&deseadoGPU, NUM_PATRONES * NUM_NEURONAS * sizeof(float));
    hipMalloc((void**)&pesosGPU, NUM_NEURONAS * NUM_ENTRADAS * sizeof(float));
    hipMalloc((void**)&pesosBiasGPU, NUM_NEURONAS * sizeof(float));
    hipMalloc((void**)&errorGPU, sizeof(int));

    // copiar desde CPU a GPU
    hipMemcpy(entradasGPU, entradasCPU, sizeof(entradasCPU), hipMemcpyHostToDevice);
    hipMemcpy(deseadoGPU, deseadoCPU, sizeof(deseadoCPU), hipMemcpyHostToDevice);

    // Inicializar pesos y bias
    hipMemset(pesosGPU, 0, NUM_NEURONAS * NUM_ENTRADAS * sizeof(float));
    hipMemset(pesosBiasGPU, 0, NUM_NEURONAS * sizeof(float));

    // ======================================================================================================
    // ENTRENAMIENTO
    int epocas = 0;
    bool bucle = false;
    while (!bucle) {
        epocas++;
        bucle = true;  // hasta encontrar un error
        for (int p = 0; p < NUM_PATRONES; ++p) {
            int cero = 0;
            hipMemcpy(errorGPU, &cero, sizeof(int), hipMemcpyHostToDevice);
            // usar el kernel para los hilos en las 10 neuronas
            entrenamientoGPU << <1, NUM_NEURONAS >> > (p, pesosGPU, pesosBiasGPU, entradasGPU, deseadoGPU, errorGPU);
            hipDeviceSynchronize();  // esperar que el kernel termine 
            
            int errorCPU;
            hipMemcpy(&errorCPU, errorGPU, sizeof(int), hipMemcpyDeviceToHost);
            if (errorCPU == 1) {
                bucle = false;  // hubo un error en la salida deseada
            }
        }
        // Terminar si hay muchas epocas
        if (epocas > 10000) {
            std::cerr << "Entrenamiento no se cumplio tras 10000 epocas.\n";
            break;
        }
    }

    // ======================================================================================================
    // RESULTADOS DE PESOS
    float pesosCPU[NUM_NEURONAS * NUM_ENTRADAS];
    hipMemcpy(pesosCPU, pesosGPU, NUM_NEURONAS * NUM_ENTRADAS * sizeof(float), hipMemcpyDeviceToHost);

    // imprimir pesos entrenados
    std::cout << "Pesos entrenados en " << epocas << " epocas.\n";
    for (int n = 0; n < NUM_NEURONAS; ++n) {
        std::cout << "Neurona " << n << " = pesos: [ ";
        for (int i = 0; i < NUM_ENTRADAS; ++i) {
            std::cout << pesosCPU[n * NUM_ENTRADAS + i] << " ";
        }
        std::cout << "]\n";
    }
    std::cout << "\n\n";


    // ======================================================================================================
    // TEST CON NUEVAS MATRICES
    const int testIndx = 1;
    float testEntradaCPU[testIndx][NUM_ENTRADAS] = {
        {1,1,1,1,1,
        0,0,0,0,1,
        0,1,1,1,1,
        0,0,0,0,1,
        1,1,1,1,1} };

    float salidasCPU[testIndx * NUM_NEURONAS];
    float* salidasGPU = nullptr;
    float* testEntradaGPU = nullptr;
    hipMalloc((void**)&salidasGPU, testIndx * NUM_NEURONAS * sizeof(float));
    hipMalloc((void**)&testEntradaGPU, testIndx * NUM_ENTRADAS * sizeof(float));
    hipMemcpy(testEntradaGPU, testEntradaCPU, sizeof(testEntradaCPU), hipMemcpyHostToDevice);

    // calcular salidas los pesos entrenados
    for (int p = 0; p < testIndx; ++p) {
        calcularSumaGPU << <1, NUM_NEURONAS >> > (p, pesosGPU, pesosBiasGPU, testEntradaGPU, salidasGPU);
    }
    hipDeviceSynchronize();
    hipMemcpy(salidasCPU, salidasGPU, testIndx * NUM_NEURONAS * sizeof(float), hipMemcpyDeviceToHost);
    
    // imprimir resultados
    for (int p = 0; p < testIndx; ++p) {
        std::cout << "Patron " << p << " = Salida: [ ";
        for (int i = 0; i < NUM_NEURONAS; ++i) {
            std::cout << salidasCPU[p * NUM_NEURONAS + i] << " ";
        }
        std::cout << "]\n";
    }

    // liberar memoria
    hipFree(testEntradaGPU);
    hipFree(entradasGPU);
    hipFree(deseadoGPU);
    hipFree(pesosGPU);
    hipFree(pesosBiasGPU);
    hipFree(errorGPU);
    hipFree(salidasGPU);

    return 0;
}
